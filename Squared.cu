
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
__global__ void cube(double* d_out, double* d_in)
{
	int idx = threadIdx.x;
	double f = d_in[idx];
	d_out[idx] = f*f*f;
	
}
int main()
{
	const uint64_t ARRAY_SIZE = 100;
	const uint64_t ARRAY_BYTES = ARRAY_SIZE * sizeof(double);

	double h_mult[ARRAY_SIZE];
	for(auto i{0};i < ARRAY_SIZE;i++)
	{
		h_mult[i] = double(i);
	}
	double h_mult_out[ARRAY_SIZE];

	double * d_in;
	double * d_out;

	hipMalloc((void **) &d_in, ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	hipMemcpy(d_in, h_mult, ARRAY_BYTES, hipMemcpyHostToDevice);
	cube<<<1, ARRAY_SIZE >>>(d_out,d_in);
	hipMemcpy(h_mult_out,d_out,ARRAY_BYTES, hipMemcpyDeviceToHost);
	for(auto i{0}; i < ARRAY_SIZE; i++)
	{
		std::cout << h_mult_out[i] << std::endl;
	}	
}
